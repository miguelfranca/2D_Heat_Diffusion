#include "hip/hip_runtime.h"
#include "evolve.h"
#include <stdio.h>
#include <algorithm>

#define BLOCK_SIZE_X 8
#define BLOCK_SIZE_Y 16
#define WORK_PER_THREAD 2

typedef struct CUDA_Config {
    CUDA_Config() {}

    float a; // Diffusion constant
    float dx; // Horizontal grid spacing
    float dy; // Vertical grid spacing
    float dx2;
    float dy2;
    float dt; // Largest stable time step

    int numElements;
    int nx, ny;

    hipStream_t comp_stream;
    hipStream_t read_stream;

    float* d_O1; // device output 1
    float* d_O2; // device output 2

    dim3 numBlocks;
    dim3 threadsPerBlock;

    bool copy;
} CUDA_Config;

CUDA_Config config;

/* Convert 2D index layout to unrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 * \param[in] array_index  The index (0 or 1) for the array position
 *
 * \returns An index in the unrolled 1D array.
 */
int __host__ __device__ getIndex(const int y, const int x, const int width,
                                 const int height,
                                 const int array_index)
{
    return (y * width + x) + (array_index * height * width);
}

__device__ void computeCurrentCell(const int x, const int y, const float* in, float* out,
                                   const int nx, const int ny, const float dx2, const float dy2,
                                   const float aTimesDt)
{
    float left = (x == 0) ? in[getIndex(y, x + 1, nx, ny, 0)] : in[getIndex(y, x - 1, nx, ny,
                 0)];

    float right = (x == nx - 1) ? in[getIndex(y, x - 1, nx, ny, 0)] : in[getIndex(y, x + 1,
                  nx,
                  nx, 0)];

    float up = (y == 0) ? in[getIndex(y + 1, x, nx, ny, 0)] : in[getIndex(y - 1, x, nx, ny,
               0)];

    float down = (y == ny - 1) ? in[getIndex(y - 1, x, nx, ny, 0)] : in[getIndex(y + 1, x, nx,
                 nx, 0)];

    float current_cell_0 = in[getIndex(y, x, nx, ny, 0)];
    float current_cell_1 = in[getIndex(y, x, nx, ny, 1)];
    //old
    // out[getIndex(y, x, nx, ny, 0)] = current_cell_0 + aTimesDt *
    // ((left - 2.0 * current_cell_0 + right) / dx2 +
    // (up - 2.0 * current_cell_0 + down) / dy2);
    // new
    out[getIndex(y, x, nx, ny, 0)] = current_cell_0 + aTimesDt * current_cell_1;
    out[getIndex(y, x, nx, ny, 1)] = current_cell_1 + aTimesDt *
                                     ((left - 2.0 * current_cell_0 + right) / dx2 +
                                      (up - 2.0 * current_cell_0 + down) / dy2);
}

__global__ void evolveKernel(const float* Un, float* Unp1, const int nx, const int ny,
                             const float dx2, const float dy2, const float aTimesDt)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    for (int w = i * WORK_PER_THREAD; w < (i + 1) * WORK_PER_THREAD ; ++w) {
        if (w < nx) { // need to check this because cuda blocks can have threads that go past the image boundaries
            int j = threadIdx.y + blockIdx.y * blockDim.y;

            if (j < ny) {
                // Explicit scheme
                computeCurrentCell(w, j, Un, Unp1, nx, ny, dx2, dy2, aTimesDt);
            }
        }
    }
}

__global__ void addHeatKernel(const int center_x, const int center_y, const float amount,
                              float* d_O1,
                              const int nx, const int ny, const float radius)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nx && y < ny) {
        int dx = x - center_x;
        int dy = y - center_y;
        int distance = dx * dx + dy * dy;

        if (distance <= radius * radius)
            d_O1[getIndex(y, x, nx, ny, 0)] = amount;
    }
}


void __host__ d_prepare(float* h_O, const int nx, const int ny)
{
    // data dimensions
    config.nx = nx;
    config.ny = ny;
    config.numElements = nx * ny;

    // initialize starting state
    float radius2 = (nx / 6.0) * (nx / 6.0);

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            int index = getIndex(i, j, ny, nx, 0);
            // Distance of point i, j from the origin
            float ds2 = (i - nx / 2) * (i - nx / 2) + (j - ny / 2) * (j - ny / 2);

            if (ds2 < radius2)
                // if (i == 0)
                h_O[index] = 100.0;
            else
                h_O[index] = 5.0;
        }
    }

    // simulation settings
    config.a = 1.0;
    config.dx = 0.005;
    config.dy = 0.005;
    config.dx2 = config.dx * config.dx;
    config.dy2 = config.dy * config.dy;
    config.dt = config.dx2 * config.dy2 / (2.0 * config.a * (config.dx2 + config.dy2));
    // config.dt = config.dx * config.dy / (2.0 * config.a * (config.dx + config.dy));

    // streams for reading and computation
    hipStreamCreate(&config.comp_stream);
    hipStreamCreate(&config.read_stream);

    // initialize memory on GPU
    hipMalloc((void**)&config.d_O1, config.numElements * 2 * sizeof(float));
    hipMalloc((void**)&config.d_O2, config.numElements * 2 * sizeof(float));

    // set PI initial values
    hipMemset(&config.d_O1[config.numElements], 0, config.numElements * sizeof(float));
    hipMemset(&config.d_O2[config.numElements], 0, config.numElements * sizeof(float));

    // copy initial state to GPU, Phi initial values
    hipMemcpy(config.d_O1, h_O, config.numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(config.d_O2, h_O, config.numElements * sizeof(float), hipMemcpyHostToDevice);

    // number of blocks and threads per block
    config.numBlocks = dim3(nx / BLOCK_SIZE_X + 1, ny / BLOCK_SIZE_Y + 1);
    config.threadsPerBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);

    config.copy = false;
}

void __host__ d_launchKernel(const int step, const int outputEvery, float* h_O)
{
    evolveKernel <<< config.numBlocks, config.threadsPerBlock, 0, config.comp_stream>>>
    (config.d_O1, config.d_O2, config.nx, config.ny, config.dx2, config.dy2,
     config.a * config.dt);

    if (config.copy) {
        int index = 0;
        hipMemcpyAsync(h_O, &config.d_O1[config.numElements * index],
                        config.numElements * sizeof(float),
                        hipMemcpyDeviceToHost,
                        config.read_stream);
        hipStreamSynchronize(config.read_stream);
        hipError_t errorCode = hipGetLastError();

        if (errorCode != hipSuccess) {
            printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
            exit(0);
        }

        config.copy = false;
    }

    if (step % outputEvery == 0) {
        config.copy = true;
        hipStreamSynchronize(config.comp_stream);
    }

    std::swap(config.d_O1, config.d_O2);
}

void __host__ d_finalize()
{
    hipStreamSynchronize(config.read_stream);
    hipStreamSynchronize(config.comp_stream);

    hipFree(config.d_O1);
    hipFree(config.d_O2);
    hipStreamDestroy(config.comp_stream);
    hipStreamDestroy(config.read_stream);
}

std::array<int, 3> scalarToRGB(double value)
{
    // Calculate the RGB values based on the hue
    // int hue = static_cast<int>(value * 240);
    int r = static_cast<int>(value * 480);
    int g = static_cast<int>(std::max(0., 240 - value * 240));
    int b = static_cast<int>(240 * (1 - value));

    // Return the RGB color as an array
    return { r, g, b };
}

void addHeat(const int x, const int y, const float amount, const float radius)
{
    addHeatKernel <<< config.numBlocks, config.threadsPerBlock, 0, config.comp_stream>>>(x, y,
            amount, config.d_O1, config.nx,
            config.ny, radius);
}

//################## Wrappers #################
void prepare(float* h_O, const int nx, const int ny)
{
    d_prepare(h_O, nx, ny);
}

void launchKernel(const int step, const int outputEvery, float* h_O)
{
    d_launchKernel(step, outputEvery, h_O);
}

void finalize()
{
    d_finalize();
}
//###############################################